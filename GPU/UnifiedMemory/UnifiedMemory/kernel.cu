#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hipblas.h>
#include <hiprand.h>

using namespace std;

void loadDimensions(int& m, int& n, int& k) {
	cout << "Give 1. matrix's row size" << endl;
	cout << "m: ";
	cin >> m;
	cout << endl << "Give 1. matrix's column size and 2. matrix's row size" << endl;
	cout << "n: ";
	cin >> n;
	cout << endl << "Matrix A dimensions: " << m << " X " << n << endl;

	cout << endl << "Give 2. matrix's column size" << endl;
	cout << "k: ";
	cin >> k;
	cout << endl << "Matrix B dimensions: " << n << " X " << k << endl;
}

void printMatrixColumnMajorOrder(float* M, int a, int b, string matrixName) {
	//print read matrix
	cout << endl << "Matrix " + matrixName + ": " << endl;

	for (int i = 0; i < a; i++) {
		for (int j = 0; j < b; j++) {
			cout << M[i + j * a] << ", ";
		}
		cout << endl;
	}
}

void randomizeMatrices(int m, int n, int k, float* A, float* B, float* C) {
	//Generating matrices

	//1. Init generator
	//arguments: (pointer to generator, type of generator to create)
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);

	//2. Set generator options (seed, offset, order)
	//arguments: (generator, seed)
	hiprandSetPseudoRandomGeneratorSeed(generator, clock());

	//3. Generate random numbers
	hiprandGenerateUniform(generator, A, m * n);
	hiprandGenerateUniform(generator, B, n * k);

	// Scaling range to [-10, 10]
	float min = -10.0f;
	float max = 10.0f;
	float scale = max - min;

	// Scale each element in d_A and d_B
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// First, multiply each element by `scale`
	hipblasSscal(handle, m * n, &scale, A, 1);
	hipblasSscal(handle, n * k, &scale, B, 1);

	// Then, add `min` to shift the values
	hipblasSaxpy(handle, m * n, &min, A, 1, A, 1);
	hipblasSaxpy(handle, n * k, &min, B, 1, B, 1);

	//4. Cleanup
	hiprandDestroyGenerator(generator);

	//input matrices are stored in column-major order
	printMatrixColumnMajorOrder(A, m, n, "A");
	printMatrixColumnMajorOrder(B, n, k, "B");
}

void fixedMatrices(int m, int n, int k, float* A, float* B) {
	//column-major order
	int number = 0;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			A[i + j * m] = number++;
		}
	}

	number = 0;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < k; j++) {
			B[i + j * n] = number++;
		}
	}

	//input matrices are stored in column-major order
	printMatrixColumnMajorOrder(A, m, n, "A");
	printMatrixColumnMajorOrder(B, n, k, "B");
}

int main() {
	//matrix dimensions
	int m, n;
	int k;

	loadDimensions(m, n, k);

	//Allocate memory
	//two matrices
	float* A, * B;

	//score matrix in GPU memory
	float* C;

	//score matrix in native memory
	float* D;

	//automatically decided if opereations should be performed on CPU or GPU
	hipMallocManaged(&A, m * n * sizeof(float));
	hipMallocManaged(&B, n * k * sizeof(float));
	hipMallocManaged(&C, m * k * sizeof(float));
	hipMallocManaged(&D, m * k * sizeof(float));

	//random matrices or fixed matrices defined by user in code
	int option;
	cout << endl << "1. Random matrices" << endl;
	cout << "2. Own matrices defined in code" << endl;
	cin >> option;

	if (option == 1) {
		randomizeMatrices(m, n, k, A, B, C);
	}
	else {
		fixedMatrices(m, n, k, A, B);
	}

	//Multiplication operation
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float alpha = 1.0f;
	float beta = 0.0f;

	//find leading dimensions of matrices
	int lda = m;
	int ldb = n;
	int ldc = m;

	//measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start event
	hipEventRecord(start, 0);

	//HIPBLAS_OP_N - non-transpose operation
	//hipblasSgemm(h,transpA,transpB,m,k,n,&alpha,&A,lda,&B,ldb,&beta,&C,ldc)
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, lda, B, ldb, &beta, C, ldc);

	//retrieve matrix from gpu memory
	//hipblasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb)
	hipblasGetMatrix(m, k, sizeof(float), C, ldc, D, ldc);

	//stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//make sure everything has finished, prevent races
	hipDeviceSynchronize();

	//start prefetching data to device, since kernel has finished
	hipMemPrefetchAsync(C, m * k, hipCpuDeviceId);

	cout << endl << "It took: " << elapsedTime << " seconds" << endl;

	//output matrix is stored in column-major order
	printMatrixColumnMajorOrder(C, m, k, "score");

	//free up memory
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);
	hipHostFree(D);
	hipblasDestroy(handle);
}