﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <random>

using namespace std;

void loadDimensions(int& m, int& n, int& k) {
	cout << "Give 1. matrix's row size" << endl;
	cout << "m: ";
	cin >> m;
	cout << endl << "Give 1. matrix's column size and 2. matrix's row size" << endl;
	cout << "n: ";
	cin >> n;
	cout << endl << "Matrix A dimensions: " << m << " X " << n << endl;

	cout << endl << "Give 2. matrix's column size" << endl;
	cout << "k: ";
	cin >> k;
	cout << endl << "Matrix B dimensions: " << n << " X " << k << endl;
}

template <typename T>
void printMatrixColumnMajorOrder(T* M, int a, int b, string matrixName) {
	//print read matrix
	cout << endl << "Matrix " + matrixName + ": " << endl;

	for (int i = 0; i < a; i++) {
		for (int j = 0; j < b; j++) {
			cout << M[i + j * a] << ", ";
		}
		cout << endl;
	}
}

template <typename T>
void randomizeMatrices(int m, int n, int k, T* A, T* B, T* C) {
	//Generating matrices

	//1. Init generator
	//arguments: (pointer to generator, type of generator to create)
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);

	//2. Set generator options (seed, offset, order)
	//arguments: (generator, seed)
	hiprandSetPseudoRandomGeneratorSeed(generator, clock());

	//3. Generate random numbers

	if constexpr (std::is_same<T, float>::value) {
		hiprandGenerateUniform(generator, A, m * n);
		hiprandGenerateUniform(generator, B, n * k);
	}
	else if constexpr (std::is_same<T, double>::value) {
		hiprandGenerateUniformDouble(generator, A, m * n);
		hiprandGenerateUniformDouble(generator, B, n * k);
	}
	else {
		// If T is something else
		std::cout << "Integer" << std::endl;
	}

	// Scaling range to [-10, 10]
	T min = -10;
	T max = 10;
	T scale = max - min;

	// Scale each element in d_A and d_B
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if constexpr (std::is_same<T, float>::value) {
		// First, multiply each element by `scale`
		hipblasSscal(handle, m * n, &scale, A, 1);
		hipblasSscal(handle, n * k, &scale, B, 1);

		// Then, add `min` to shift the values
		hipblasSaxpy(handle, m * n, &min, A, 1, A, 1);
		hipblasSaxpy(handle, n * k, &min, B, 1, B, 1);
	}
	else if constexpr (std::is_same<T, double>::value) {
		// First, multiply each element by `scale`
		hipblasDscal(handle, m * n, &scale, A, 1);
		hipblasDscal(handle, n * k, &scale, B, 1);

		// Then, add `min` to shift the values
		hipblasDaxpy(handle, m * n, &min, A, 1, A, 1);
		hipblasDaxpy(handle, n * k, &min, B, 1, B, 1);
	}
	else {
		//For integers, there's no method to generate numbers in cublas library, so classicaly:
		for (int i = 0; i < m * n; i++) {
			A[i] = (rand() % (max - min + 1)) + min;
		}

		for (int i = 0; i < n * k; i++) {
			B[i] = (rand() % (max - min + 1)) + min;
		}
	}

	//4. Cleanup
	hiprandDestroyGenerator(generator);

	//input matrices are stored in column-major order
	printMatrixColumnMajorOrder(A, m, n, "A");
	printMatrixColumnMajorOrder(B, n, k, "B");
}

template <typename T>
void fixedMatrices(int m, int n, int k, T* A, T* B) {
	//column-major order
	int number = 0;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			A[i + j * m] = number++;
		}
	}

	number = 0;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < k; j++) {
			B[i + j * n] = number++;
		}
	}

	//input matrices are stored in column-major order
	printMatrixColumnMajorOrder(A, m, n, "A");
	printMatrixColumnMajorOrder(B, n, k, "B");
}

//generic
template <typename T>
void program() {
	//matrix dimensions
	int m, n;
	int k;

	loadDimensions(m, n, k);

	//Allocate memory
	//two matrices
	T* A, * B;

	//score matrix in GPU memory
	T* C;

	//score matrix in native memory
	T* D;

	hipHostMalloc(&A, m * n * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	hipHostMalloc(&B, n * k * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	hipHostMalloc(&C, m * k * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	hipHostMalloc(&D, m * k * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);

	//random matrices or fixed matrices defined by user in code
	int option;
	cout << endl << "1. Random matrices" << endl;
	cout << "2. Own matrices defined in code" << endl;
	cin >> option;

	if (option == 1) {
		randomizeMatrices<T>(m, n, k, A, B, C);
	}
	else {
		fixedMatrices<T>(m, n, k, A, B);
	}

	//Multiplication operation
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	T alpha = 1;
	T beta = 0;

	//find leading dimensions of matrices
	int lda = m;
	int ldb = n;
	int ldc = m;

	//measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start event
	hipEventRecord(start, 0);

	//CUBLAS_OP_N - non-transpose operation
	//cublasSgemm(h,transpA,transpB,m,k,n,&alpha,&A,lda,&B,ldb,&beta,&C,ldc)

	if constexpr (std::is_same<T, float>::value) {
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, lda, B, ldb, &beta, C, ldc);
	}
	else if constexpr (std::is_same<T, double>::value) {
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, lda, B, ldb, &beta, C, ldc);
	}
	else {
		hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, HIP_R_8I, lda, B, HIP_R_8I, ldb, &beta, C, HIP_R_8I, ldc, HIP_R_8I, HIPBLAS_GEMM_DEFAULT);
	}

	//retrieve matrix from gpu memory
	//cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb)
	hipblasGetMatrix(m, k, sizeof(T), C, ldc, D, ldc);

	//stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << endl << "It took: " << elapsedTime << " seconds" << endl;

	//output matrix is stored in column-major order
	printMatrixColumnMajorOrder<T>(C, m, k, "score");

	//free up memory
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);
	hipHostFree(D);
	hipblasDestroy(handle);
}

int main() {
	int dataType;
	cout << "Choose data type: " << endl;
	cout << "1. Float" << endl;
	cout << "2. Integer" << endl;
	cout << "3. Double" << endl;
	cin >> dataType;

	switch (dataType) {
	case 1: program<float>(); break;
	case 2: program<int>(); break;
	case 3: program<double>(); break;
	}
}