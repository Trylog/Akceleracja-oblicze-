﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <hiprand/hiprand.h>

using namespace std;

void loadDimensions(int& m, int& n, int& k) {
	cout << "Give 1. matrix's row size" << endl;
	cout << "m: ";
	cin >> m;
	cout << endl << "Give 1. matrix's column size and 2. matrix's row size" << endl;
	cout << "n: ";
	cin >> n;
	cout << endl << "Matrix A dimensions: " << m << " X " << n << endl;

	cout << endl << "Give 2. matrix's column size" << endl;
	cout << "k: ";
	cin >> k;
	cout << endl << "Matrix B dimensions: " << n << " X " << k << endl;
}

void printMatrix(float* M, int a, int b, string matrixName) {
	//print read matrix
	cout << endl << "Matrix " + matrixName + ": " << endl;
	for (int i = 0; i < a; i++) {
		for (int j = 0; j < b; j++) {
			cout << M[i * b + j] << ", ";
		}
		cout << endl;
	}
}

void randomizeMatrices(int m, int n, int k, float* A, float* B, float* C) {
	//Generating matrices

	//1. Init generator
	//arguments: (pointer to generator, type of generator to create)
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);

	//2. Set generator options (seed, offset, order)
	//arguments: (generator, seed)
	hiprandSetPseudoRandomGeneratorSeed(generator, clock());

	//3. Generate random numbers
	hiprandGenerateUniform(generator, A, m * n);
	hiprandGenerateUniform(generator, B, n * k);

	// Scaling range to [-10, 10]
	float min = -10.0f;
	float max = 10.0f;
	float scale = max - min;

	// Scale each element in d_A and d_B
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// First, multiply each element by `scale`
	hipblasSscal(handle, m * n, &scale, A, 1);
	hipblasSscal(handle, n * k, &scale, B, 1);

	// Then, add `min` to shift the values
	hipblasSaxpy(handle, m * n, &min, A, 1, A, 1);
	hipblasSaxpy(handle, n * k, &min, B, 1, B, 1);

	//4. Cleanup
	hiprandDestroyGenerator(generator);

	printMatrix(A, m, n, "A");
	printMatrix(B, n, k, "B");
}

void fixedMatrices(int m, int n, int k, float* A, float* B) {
	//row-major order
	for (int i = 0; i < m * n; i++) {
		A[i] = i;
	}

	for (int i = 0; i < n * k; i++) {
		B[i] = i;
	}

	printMatrix(A, m, n, "A");
	printMatrix(B, n, k, "B");
}

int main() {
	//matrix dimensions
	int m, n;
	int k;

	loadDimensions(m, n, k);

	//Allocate memory
	//two matrices
	float* A, * B;

	//score matrix in GPU memory
	float* C;

	//score matrix in native memory
	float* D;

	hipHostMalloc(&A, m * n * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&B, n * k * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&C, m * k * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&D, m * k * sizeof(float), hipHostMallocDefault);

	//random matrices or fixed matrices defined by user in code
	int option;
	cout << endl << "1. Random matrices" << endl;
	cout << "2. Own matrices defined in code" << endl;
	cin >> option;

	if (option == 1) {
		randomizeMatrices(m, n, k, A, B, C);
	}
	else {
		fixedMatrices(m, n, k, A, B);
	}

	//Multiplication operation
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float alpha = 1.0f;
	float beta = 0.0f;

	//find leading dimensions of matrices
	int lda = m;
	int ldb = n;
	int ldc = m;

	//measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start event
	hipEventRecord(start, 0);

	//CUBLAS_OP_N - non-transpose operation
	//cublasSgemm(h,transpA,transpB,m,k,n,&alpha,&A,lda,&B,ldb,&beta,&C,ldc)
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, B, ldb, A, lda, &beta, C, ldc);

	//retrieve matrix from gpu memory
	//cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb)
	hipblasGetMatrix(m, k, sizeof(float), C, ldc, D, ldc);

	//stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << endl << "It took: " << elapsedTime << " seconds" << endl;

	printMatrix(D, m, k, "score");

	//free up memory
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);
	hipHostFree(D);
	hipblasDestroy(handle);
}