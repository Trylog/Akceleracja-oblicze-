﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <random>

using namespace std;

void loadDimensions(int& m, int& n, int& k) {
	cout << "Give 1. matrix's row size" << endl;
	cout << "m: ";
	cin >> m;
	cout << endl << "Give 1. matrix's column size and 2. matrix's row size" << endl;
	cout << "n: ";
	cin >> n;
	cout << endl << "Matrix A dimensions: " << m << " X " << n << endl;

	cout << endl << "Give 2. matrix's column size" << endl;
	cout << "k: ";
	cin >> k;
	cout << endl << "Matrix B dimensions: " << n << " X " << k << endl;
}

template <typename T>
void printMatrixColumnMajorOrder(T* M, int a, int b, string matrixName) {
	//print read matrix
	cout << endl << "Matrix " + matrixName + ": " << endl;

	for (int i = 0; i < a; i++) {
		for (int j = 0; j < b; j++) {
			if (std::is_same<T, int8_t>::value) {
				cout << static_cast<int32_t>(M[i + j * a]);
			}
			else {
				cout << M[i + j * a];
			}
			if (j < b - 1) {
				cout << ", ";
			}
		}
		cout << endl;
	}
}

template <typename T>
void randomizeMatrices(int m, int n, int k, T* A, T* B) {
	
	//Generating matrices
	//float, double
	if constexpr (std::is_same<T, float>::value || std::is_same<T, double>::value) {
		//1. Init generator
		//arguments: (pointer to generator, type of generator to create)
		hiprandGenerator_t generator;
		hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);

		//2. Set generator options (seed, offset, order)
		//arguments: (generator, seed)
		hiprandSetPseudoRandomGeneratorSeed(generator, clock());

		//3. Generate random numbers
		//float
		if constexpr (std::is_same<T, float>::value) {
			hiprandGenerateUniform(generator, A, m * n);
			hiprandGenerateUniform(generator, B, n * k);
		}
		//double
		else {
			hiprandGenerateUniformDouble(generator, A, m * n);
			hiprandGenerateUniformDouble(generator, B, n * k);
		}

		// Scaling range to [-10, 10]
		T min = -128;
		T max = 127;
		T scale = max - min;

		// Scale each element in d_A and d_B
		hipblasHandle_t handle;
		hipblasCreate(&handle);

		//float
		if constexpr (std::is_same<T, float>::value) {
			// First, multiply each element by `scale`
			hipblasSscal(handle, m * n, &scale, A, 1);
			hipblasSscal(handle, n * k, &scale, B, 1);

			// Then, add `min` to shift the values
			hipblasSaxpy(handle, m * n, &min, A, 1, A, 1);
			hipblasSaxpy(handle, n * k, &min, B, 1, B, 1);
		}
		//double
		else {
			// First, multiply each element by `scale`
			hipblasDscal(handle, m * n, &scale, A, 1);
			hipblasDscal(handle, n * k, &scale, B, 1);

			// Then, add `min` to shift the values
			hipblasDaxpy(handle, m * n, &min, A, 1, A, 1);
			hipblasDaxpy(handle, n * k, &min, B, 1, B, 1);
		}

		//4. Cleanup
		hiprandDestroyGenerator(generator);

		//input matrices are stored in column-major order
		printMatrixColumnMajorOrder(A, m, n, "A");
		printMatrixColumnMajorOrder(B, n, k, "B");
	}

	//For integers, there's no method to generate numbers in curand library, so classicaly:
	else {
		srand(static_cast<unsigned int>(time(NULL)));
		//matrix dimensions have to be multiplication of 4
		int closestM = ((m + 3) / 4) * 4;
		int closestN = ((n + 3) / 4) * 4;

		for (int i = 0; i < closestM; i++) {
			for (int j = 0; j < closestN; j++) {
				if (i >= m || j >= n) {
					A[i * closestM + j] = 0;
				}
				else {
					A[i * closestM + j] = static_cast<T>(rand() % (256 - 128));
				}
			}
		}

		int closestK = ((k + 3) / 4) * 4;
		for (int i = 0; i < closestN; i++) {
			for (int j = 0; j < closestK; j++) {
				if (i >= n || j >= k) {
					B[i * closestN + j] = 0;
				}
				else {
					B[i * closestN + j] = static_cast<T>(rand() % (256 - 128));
				}
			}
		}
		//input matrices are stored in column-major order
		printMatrixColumnMajorOrder(A, closestM, closestN, "A");
		printMatrixColumnMajorOrder(B, closestN, closestK, "B");
	}
}

int* removePadding(int* M, int closestM, int closestK, int m, int k) {
	int* score = new int[m * k];

	for (int i = 0; i < closestM; i++) {
		for (int j = 0; j < closestK; j++) {
			if (i < m || j < k) {
				score[i * closestM + j] = M[i * closestM + j];
			}
		}
	}

	return score;
}

template <typename T, typename U>
void distinguish(int m, int n, int k, T* A, T* B) {
	U* C;
	//score matrix in native memory
	U* D;

	if (std::is_same<T, double>::value || std::is_same<T, float>::value) {
		hipHostMalloc(&C, m * k * sizeof(U), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
		hipHostMalloc(&D, m * k * sizeof(U), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	}
	else {
		int closestM = ((m + 3) / 4) * 4;
		int closestK = ((k + 3) / 4) * 4;

		hipHostMalloc(&C, closestM * closestK * sizeof(U), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
		hipHostMalloc(&D, closestM * closestK * sizeof(U), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	}
	

	randomizeMatrices<T>(m, n, k, A, B);

	//Multiplication operation
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	U alpha = 1;
	U beta = 0;

	//CUBLAS_OP_N - non-transpose operation
	//cublasSgemm(h,transpA,transpB,m,k,n,&alpha,&A,lda,&B,ldb,&beta,&C,ldc)

	if constexpr (std::is_same<U, float>::value || std::is_same<U, double>::value) {
		//find leading dimensions of matrices
		int lda = m;
		int ldb = n;
		int ldc = m;

		//measure time
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		//start event
		hipEventRecord(start, 0);

		//float
		if constexpr (std::is_same<U, float>::value) {
			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, lda, B, ldb, &beta, C, ldc);
		}

		//double
		else {
			hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, lda, B, ldb, &beta, C, ldc);
		}

		//retrieve matrix from gpu memory
		//cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb)
		hipblasGetMatrix(m, k, sizeof(U), C, ldc, D, ldc);

		//stop event
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		cout << endl << "It took: " << elapsedTime << " milliseconds" << endl;

		//output matrix is stored in column-major order
		printMatrixColumnMajorOrder<U>(C, m, k, "score");
	}

	else {
		int closestM = ((m + 3) / 4) * 4;
		int closestN = ((n + 3) / 4) * 4;

		int closestK = ((k + 3) / 4) * 4;

		//find leading dimensions of matrices
		int lda = closestM;
		int ldb = closestN;
		int ldc = closestM;

		//measure time
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		//start event
		hipEventRecord(start, 0);
		hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, closestM, closestK, closestN, &alpha, A, HIP_R_8I, lda, B, HIP_R_8I, ldb, &beta, C, HIP_R_32I, ldc, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);

		//retrieve matrix from gpu memory
		//cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb)
		hipblasGetMatrix(closestM, closestK, sizeof(U), C, ldc, D, ldc);

		//stop event
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		cout << endl << "It took: " << elapsedTime << " milliseconds" << endl;

		//output matrix is stored in column-major order
		int * score = removePadding(C, closestM, closestK, m, k);
		printMatrixColumnMajorOrder<U>(score, m, k, "score");
	}

	//free up memory
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);
	hipHostFree(D);
	hipblasDestroy(handle);
}

//generic
template <typename T>
void program() {
	//matrix dimensions
	int m, n;
	int k;

	loadDimensions(m, n, k);

	//Allocate memory
	//two matrices
	T* A, * B;

	if (std::is_same<T, double>::value || std::is_same<T, float>::value) {
		hipHostMalloc(&A, m * n * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
		hipHostMalloc(&B, n * k * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	}
	//int8_t, dimensions have to be multiplication of 4
	else {
		int closestM = ((m + 3) / 4) * 4;
		int closestN = ((n + 3) / 4) * 4;

		int closestK = ((k + 3) / 4) * 4;

		hipHostMalloc(&A, closestM * closestN * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
		hipHostMalloc(&B, closestN * closestK * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
	}
	

	//score matrix in GPU memory
	//for 8bit integers - score is 32 bits
	if constexpr (std::is_same<T, int8_t>::value) {
		distinguish<T, int32_t>(m, n, k, A, B);
	}
	else {
		distinguish<T, T>(m, n, k, A, B);
	}
}

int main() {
	int dataType;
	cout << "Choose data type: " << endl;
	cout << "1. Float" << endl;
	cout << "2. Integer" << endl;
	cout << "3. Double" << endl;
	cin >> dataType;

	switch (dataType) {
	case 1: program<float>(); break;
	case 2: program<int8_t>(); break;
	case 3: program<double>(); break;
	}
}