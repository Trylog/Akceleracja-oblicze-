﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <hiprand/hiprand.h>

using namespace std;

void loadDimensions(int& m, int& n, int& k) {
	cout << "Give 1. matrix's row size" << endl;
	cout << "m: ";
	cin >> m;
	cout << endl << "Give 1. matrix's column size and 2. matrix's row size" << endl;
	cout << "n: ";
	cin >> n;
	cout << endl << "Matrix A dimensions: " << m << " X " << n << endl;

	cout << endl << "Give 2. matrix's column size" << endl;
	cout << "k: ";
	cin >> k;
	cout << endl << "Matrix B dimensions: " << n << " X " << k << endl;
}

void randomizeMatrices(int m, int n, int k, float* A, float* B, float* C) {
	//Generating matrices

	//1. Init generator
	//arguments: (pointer to generator, type of generator to create)
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);

	//2. Set generator options (seed, offset, order)
	//arguments: (generator, seed)
	hiprandSetPseudoRandomGeneratorSeed(generator, 1234UL);

	//3. Generate random numbers
	hiprandGenerateUniform(generator, A, m * n);
	hiprandGenerateUniform(generator, B, n * k);

	// Scaling range to [-10, 10]
	float min = -10.0f;
	float max = 10.0f;
	float scale = max - min;

	// Scale each element in d_A and d_B
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// First, multiply each element by `scale`
	hipblasSscal(handle, m * n, &scale, A, 1);
	hipblasSscal(handle, n * k, &scale, B, 1);

	// Then, add `min` to shift the values
	hipblasSaxpy(handle, m * n, &min, A, 1, A, 1);
	hipblasSaxpy(handle, n * k, &min, B, 1, B, 1);

	//4. Cleanup
	hiprandDestroyGenerator(generator);


	//Check matrices which were generated
	float* generatedMatrixA = (float*)malloc(m * n * sizeof(float));
	float* generatedMatrixB = (float*)malloc(n * k * sizeof(float));

	hipMemcpy(generatedMatrixA, A, m * n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(generatedMatrixB, B, n * k * sizeof(float), hipMemcpyDeviceToHost);

	//print read matrix
	cout << endl << "Matrix A:" << endl;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			cout << A[i * n + j] << ", ";
		}
		cout << endl;
	}

	//print read matrix
	cout << endl << "Matrix B:" << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < k; j++) {
			cout << B[i * k + j] << ", ";
		}
		cout << endl;
	}
}

void fixedMatrices(int m, int n, int k, float* A, float* B) {
	//row-major order
	for (int i = 0; i < m * n; i++) {
		A[i] = i;
	}

	for (int i = 0; i < n * k; i++) {
		B[i] = i;
	}

	//print read matrix
	cout << endl << "Matrix A:" << endl;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			cout << A[i * n + j] << ", ";
		}
		cout << endl;
	}

	//print read matrix
	cout << endl << "Matrix B:" << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < k; j++) {
			cout << B[i * k + j] << ", ";
		}
		cout << endl;
	}
}

int main() {

	//matrix dimensions
	int m, n;
	int k;

	loadDimensions(m, n, k);

	//Allocate memory
	//two matrices
	float* A, * B;

	//score matrix in GPU memory
	float* C;

	//score matrix in native memory
	float* D;

	hipHostMalloc(&A, m * n * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&B, n * k * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&C, m * k * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&D, m * k * sizeof(float), hipHostMallocDefault);

	//random matrices or fixed matrices defined by user in code
	int option;
	cout << endl << "1. Random matrices" << endl;
	cout << "2. Own matrices defined in code" << endl;
	cin >> option;

	if (option == 1) {
		randomizeMatrices(m, n, k, A, B, C);
	}
	else {
		fixedMatrices(m, n, k, A, B);
	}

	//Multiplication operation
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float alpha = 1.0f;
	float beta = 0.0f;

	//find leading dimensions of matrices
	int lda = m;
	int ldb = n;
	int ldc = m;

	//measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start event
	hipEventRecord(start, 0);

	//CUBLAS_OP_N - non-transpose operation
	//cublasSgemm(h,transpA,transpB,m,k,n,&alpha,&A,lda,&B,ldb,&beta,&C,ldc)
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A, lda, B, ldb, &beta, C, ldc);

	//retrieve matrix from gpu memory
	//cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb)
	hipblasGetMatrix(m, k, sizeof(float), C, ldc, D, ldc);

	//stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << endl << "It took: " << elapsedTime << " seconds" << endl;

	//print score matrix
	//row-major order
	cout << "Score matrix:" << endl;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < k; j++) {
			cout << D[i * k + j] << ", ";
		}
		cout << endl;
	}
}